#include "hip/hip_runtime.h"
#include "includes/block_reduce.h"
#include "includes/kernels.h"
#include "includes/cuda_util.h"

#include <hip/hip_cooperative_groups.h>
#include <cstddef>

namespace cg = cooperative_groups;
namespace lightseq {
namespace cuda {

const float LN_EPSILON = 1e-8f;
#define TILE_DIM 32


/**
@brief: ker_layer_norm
Standard layer normalization.
It will not only output the layer norm result,
  but also outputs variance.
  may also output means, depends on whether
  the means argument is nullptr

@thread
gridDim.x = batch_size * seq_len
blockDim.x = hidden_size

@param
ln_res: [batch_size * seq_len, hidden_size], ln result.
vars: [batch_size * seq_len], variance per token
means: [batch_size * seq_len], means per token, can be nullput
inp: [batch_size * seq_len, hidden_size], ln input.
scale: [hidden_size], ln scale
bias: [hidden_size], ln bias
*/
template <typename T>
__global__ void ker_layer_norm(T *ln_res, T *vars, T *means, const T *inp,
                               const T *scale, const T *bias, int hidden_size) {
  
  /// BEGIN ASSIGN4_2_1
  /// TODO
  // Hints:
  // 1. Compute x and x^2 with reinterpret_cast by casting to float4 for speedup
  // 2. Compute reduce sum with blockReduce and add epsilon with LN_EPSILON
  // 3. Compute layernorm result with reinterpret_cast by casting to float4 for speedup
  
  // Step 1
  float l_sum = 0;
  float l_sum_sq = 0;
  float sum_values[2];
  const float4 *inp_f4 = reinterpret_cast<const float4 *>(inp) + blockIdx.x * hidden_size;  
  for (uint idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float4 val = inp_f4[idx];
    l_sum += val.x + val.y + val.z + val.w;
    l_sum_sq += (val.x * val.x) + (val.y * val.y) + (val.z * val.z) + (val.w * val.w);
  }
  sum_values[0] = l_sum;
  sum_values[1] = l_sum_sq;
  
  // Step 2
  blockReduce<ReduceType::kSum, 2>(sum_values);
  
  // write shared
  __shared__ float mean;
  __shared__ float mean_of_squares;
  __shared__ float var;

  if (threadIdx.x == 0) {
    l_sum = sum_values[0];
    l_sum_sq = sum_values[1];
    mean = __fdividef(l_sum, hidden_size*4);
    mean_of_squares = __fdividef(l_sum_sq, hidden_size*4);
    var = (mean_of_squares - (mean * mean)) + LN_EPSILON;

    if (means) means[blockIdx.x] = mean;
    vars[blockIdx.x] = var;
  }
  __syncthreads();
  // Step 3

  float4 *ln_res_f4 = reinterpret_cast<float4 *>(ln_res) + blockIdx.x * (hidden_size); 
  const float4 *scale_f4 = reinterpret_cast<const float4 *>(scale); 
  const float4 *bias_f4 = reinterpret_cast<const float4 *>(bias); 
  for (uint idx = threadIdx.x; idx < hidden_size; idx += blockDim.x) {
    float4 val = inp_f4[idx];
    float4 scale_val = scale_f4[idx];
    float4 bias_val = bias_f4[idx];
    float4 result;
    result.x = ((val.x - mean) / sqrtf(var)) * scale_val.x + bias_val.x;
    result.y = ((val.y - mean) / sqrtf(var)) * scale_val.y + bias_val.y;
    result.z = ((val.z - mean) / sqrtf(var)) * scale_val.z + bias_val.z;
    result.w = ((val.w - mean) / sqrtf(var)) * scale_val.w + bias_val.w;
    ln_res_f4[idx] = result;
  }
  /// END ASSIGN4_2_1
}

extern "C" {
void launch_layernorm(float *ln_res, float *vars, float *means,
                              const float *inp, const float *scale,
                              const float *bias, int batch_size, int hidden_dim,
                              hipStream_t stream) {
  if (hidden_dim % 4 != 0) {
    throw std::runtime_error("violate hidden_dim % 4 = 0");
  }
  int float_size = sizeof(float);
  int input_size = batch_size * hidden_dim * float_size;
  int scale_size = hidden_dim * float_size;
  int bias_size = hidden_dim * float_size;
  int output_size = batch_size * hidden_dim * float_size;
  int mean_size = batch_size * float_size;
  int var_size = batch_size * float_size;


  float *d_ln_res, *d_vars, *d_means, *d_inp, *d_scale, *d_bias;
  hipMalloc((void **)&d_ln_res, output_size);
  hipMalloc((void **)&d_vars, var_size);
  hipMalloc((void **)&d_means, mean_size);
  hipMalloc((void **)&d_inp, input_size);
  hipMalloc((void **)&d_scale, scale_size);
  hipMalloc((void **)&d_bias, bias_size);

  hipMemcpy(d_inp, inp, input_size, hipMemcpyHostToDevice);
  hipMemcpy(d_scale, scale, scale_size, hipMemcpyHostToDevice);
  hipMemcpy(d_bias, bias, bias_size, hipMemcpyHostToDevice);

  // For using float4
  hidden_dim >>= 2;
  int nthread = min(((hidden_dim + 31) / 32) * 32, MAX_THREADS);
  dim3 grid_dim(batch_size);
  dim3 block_dim(nthread);

  ker_layer_norm<float><<<grid_dim, block_dim, 0, stream>>>(
      d_ln_res, d_vars, d_means, d_inp, d_scale, d_bias, hidden_dim);

  // Copy back to the host
  hipMemcpy(ln_res, d_ln_res, output_size, hipMemcpyDeviceToHost);
  hipMemcpy(vars, d_vars, var_size, hipMemcpyDeviceToHost);
  hipMemcpy(means, d_means, mean_size, hipMemcpyDeviceToHost);
  hipDeviceSynchronize();

  // Check CUDA execution
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "launch_layernorm Error: %s\n", hipGetErrorString(err));
    // Handle the error (e.g., by exiting the program)
    exit(EXIT_FAILURE);
  }

  // Free memory on device
  hipFree(d_ln_res);
  hipFree(d_vars);
  hipFree(d_means);
  hipFree(d_inp);
  hipFree(d_scale);
  hipFree(d_bias);

}
}

/**
@brief: ker_ln_bw_dgamma_dbetta
Layer norm backword kernel, compute the gradient of gamma and betta.
dbetta = sum(dout, dim=0)
dgamma = sum(xhat * dout, dim=0)
xhat = (input - mean) * rsqrt(var) or
  (output - betta) / gamma

@thread
gridDim.x = hidden_size / 32
blockDim.x = 32
blockDim.y = 32

@param
gamma_grad: [hidden_size], gradient of gamma
betta_grad: [hidden_size], gradient of betta
out_grad: [batch_size * seq_len, hidden_size], gradient of betta ln output
inp_or_out: [batch_size * seq_len, hidden_size], ln output if means is nullptr
  ln input if means is not nullptr
gamma: [hidden_size], gamma of ln,
  used to compute xhat, maybe nullptr
betta: [hidden_size], betta of ln,
  used to compute xhat, maybe nullptr
vars: [batch_size * seq_len], variance of ln forward,
  used to compute xhat, maybe nullptr
means: [batch_size * seq_len], mean of ln forward,
  used to compute xhat, maybe nullptr
(gamma && betta) ^ (vars && means) should be true
*/
template <typename T>
__global__ void ker_ln_bw_dgamma_dbetta(T *gamma_grad, T *betta_grad,
                                        const T *out_grad,
                                        const T *inp, const T *gamma,
                                        const T *betta, const T *vars,
                                        const T *means, int rows, int width) {

  /// BEGIN ASSIGN4_2_2
  /// TODO
  // Hints:
  // 1. Compute the partial gradients by looping across inp rows
  // 2. Store the partial gradients in the shared memory arrays
  // 3. Compute the reduce sum of the shared memory arrays with g.shfl_down
  //      -> More hints about `g.shfl_down`:
  //      -> https://developer.nvidia.com/blog/cooperative-groups/#:~:text=Using%20thread_block_tile%3A%3Ashfl_down()%20to%20simplify%20our%20warp%2Dlevel%20reduction%20does%20benefit%20our%20code%3A%20it%20simplifies%20it%20and%20eliminates%20the%20need%20for%20shared%20memory
  //      -> The highlighted line gives you a conceptual understanding of what the g.shfl_down is doing. Usually, the threads inside a block need to load everything to shared memory and work together to reduce the result (like what you have implemented in the hw1 for reduce function). 
  //      -> Now g.shfl_down helps you do so without consuming any shared memory. g.shfl_down makes it more efficient.
  // 4. Assign the final result to the correct position in the global output

  __shared__ float betta_buffer[TILE_DIM][TILE_DIM];
  __shared__ float gamma_buffer[TILE_DIM][TILE_DIM];

  cg::thread_block b = cg::this_thread_block();
  cg::thread_block_tile<TILE_DIM> g = cg::tiled_partition<TILE_DIM>(b);

  
  // Step 1
  float local_betta_grad = 0.0f;
  float local_gamma_grad = 0.0f;
  int col = threadIdx.x + blockIdx.x * blockDim.x;

  for (uint row = threadIdx.y; row < rows; row += blockDim.y) {
    if (col < width) {
      float inp_val = inp[row * width + col];
      float out_grad_val = out_grad[row * width + col];

      float xhat;
      if (means){
        float vars_val = vars[row];
        float means_val = means[row];
        xhat = (inp_val - means_val) * rsqrtf(vars_val);
      } else {
        float gamma_val = gamma[col];
        float betta_val = betta[col];
        xhat = (inp_val - betta_val) / gamma_val;
      }

      local_betta_grad += out_grad_val;
      local_gamma_grad += out_grad_val * xhat;
    }
  }
  // Step 2
  betta_buffer[threadIdx.y][threadIdx.x] = local_betta_grad;
  gamma_buffer[threadIdx.y][threadIdx.x] = local_gamma_grad;
  __syncthreads();

  // Step 3
  if (threadIdx.y == 0) {
    for (int k = 0; k < TILE_DIM; ++k) {
      int out_col = blockIdx.x * TILE_DIM + k;
      if (out_col >= width) break;
  
      float sum_beta  = betta_buffer[threadIdx.x][k]; 
      float sum_gamma = gamma_buffer[threadIdx.x][k];
  
      // sums over rows
      for (int offset = TILE_DIM / 2; offset > 0; offset >>= 1) {
        sum_beta  += g.shfl_down(sum_beta,  offset);
        sum_gamma += g.shfl_down(sum_gamma, offset);
      }
  
      // Step 4

      if (threadIdx.x == 0) {
        betta_grad[out_col] = sum_beta;
        gamma_grad[out_col] = sum_gamma;
      }
    }
  }

  /// END ASSIGN4_2_2
}

/**
@brief: ker_ln_bw_dinp
Layer norm backword kernel, compute the gradient of input.
dinp = (dxhat - (sum(dxhat) + xhat * sum(dxhat * xhat)) / hidden_dim)
  * rsqrt(var)
xhat = (input - mean) * rsqrt(var) if mean is not nullptr
       (output - betta) / gamma if mean is nullptr
dxhat = dout * gamma


@thread
gridDim.x = batch_size * seq_len
blockDim.x = hidden_size

@param
inp_grad: [batch_size * seq_len, hidden_size], gradient of betta ln output
out_grad: [batch_size * seq_len, hidden_size], gradient of betta ln output
residual_grad: [batch_size * seq_len, hidden_size], gradient of residual input,
  usually appear in pre-layer-norm for transformer layer, maybe nullptr
inp_or_out: [batch_size * seq_len, hidden_size], ln output if means is nullptr
  ln input if means is not nullptr
gamma: [hidden_size], gamma of ln,
  used to compute xhat and dxhat
betta: [hidden_size], betta of ln,
  used to compute xhat, maybe nullptr
vars: [batch_size * seq_len], variance of ln forward,
  used to compute xhat and dinp
means: [batch_size * seq_len], mean of ln forward,
  used to compute xhat, maybe nullptr
*/
template <typename T>
__global__ void ker_ln_bw_dinp(T *inp_grad, const T *out_grad, const T *inp,
                               const T *gamma, const T *betta, const T *vars,
                               const T *means, int hidden_dim) {
  
  /// BEGIN ASSIGN4_2_2
  /// TODO
  // Hints:
  // 1. Compute dxhat=dy*w with reinterpret_cast by casting to float4 for speedup
  // 2. Compute xhat with reinterpret_cast by casting to float4 for speedup
  // 3. Compute reduce sum for dxhat and dxhat*xhat with blockReduce
  // 4. Compute final gradient
  
  // Step 1
  const float4 *out_grad_f4 = reinterpret_cast<const float4 *>(out_grad) + blockIdx.x * (hidden_dim); 
  float4 *inp_grad_f4 = reinterpret_cast<float4 *>(inp_grad) + blockIdx.x * (hidden_dim); 
  const float4 *inp_f4 = reinterpret_cast<const float4 *>(inp) + blockIdx.x * (hidden_dim);
  const float4 *gamma_f4 = reinterpret_cast<const float4 *>(gamma);
  const float4 *betta_f4 = reinterpret_cast<const float4 *>(betta);
  float sum_dxhat = 0.0f;
  float sum_dxhat_xhat = 0.0f; 
  float sum_values[2];
  // Step 2
   
  for (uint idx = threadIdx.x; idx < hidden_dim; idx += blockDim.x) {
    float4 inp_val = inp_f4[idx];
    float4 xhat_f4;
    float4 dxhat_f4;
    float4 betta_val = betta_f4[idx];
    float4 gamma_val = gamma_f4[idx];
    float4 out_grad_val = out_grad_f4[idx];
 
    if (means){
      xhat_f4.x = (inp_val.x - means[blockIdx.x]) * rsqrtf(vars[blockIdx.x]);
      xhat_f4.y = (inp_val.y - means[blockIdx.x]) * rsqrtf(vars[blockIdx.x]);
      xhat_f4.z = (inp_val.z - means[blockIdx.x]) * rsqrtf(vars[blockIdx.x]);
      xhat_f4.w = (inp_val.w - means[blockIdx.x]) * rsqrtf(vars[blockIdx.x]);

    } else {
      xhat_f4.x = (inp_val.x - betta_val.x) / gamma_val.x;
      xhat_f4.y = (inp_val.y - betta_val.y) / gamma_val.y;
      xhat_f4.z = (inp_val.z - betta_val.z) / gamma_val.z;
      xhat_f4.w = (inp_val.w - betta_val.w) / gamma_val.w;
    }

    dxhat_f4.x = out_grad_val.x * gamma_val.x; 
    dxhat_f4.y = out_grad_val.y * gamma_val.y; 
    dxhat_f4.z = out_grad_val.z * gamma_val.z; 
    dxhat_f4.w = out_grad_val.w * gamma_val.w; 

    sum_dxhat += dxhat_f4.x + dxhat_f4.y + dxhat_f4.z + dxhat_f4.w;
    sum_dxhat_xhat += dxhat_f4.x * xhat_f4.x + dxhat_f4.y * xhat_f4.y + dxhat_f4.z * xhat_f4.z + dxhat_f4.w * xhat_f4.w;
  }
  sum_values[0] = sum_dxhat;
  sum_values[1] = sum_dxhat_xhat;
  
  // Step 3
  blockReduce<ReduceType::kSum, 2>(sum_values);
  __shared__ float s_sum_dxhat;
  __shared__ float s_sum_dxhat_xhat;
  if (threadIdx.x == 0) {
    s_sum_dxhat = sum_values[0];
    s_sum_dxhat_xhat = sum_values[1];
  }
  __syncthreads();

  // Step 4
  for (uint idx = threadIdx.x; idx < hidden_dim; idx += blockDim.x) {
    float4 inp_val = inp_f4[idx];
    float4 xhat_f4;
    float4 dxhat_f4;
    float4 betta_val = betta_f4[idx];
    float4 gamma_val = gamma_f4[idx];
    float4 out_grad_val = out_grad_f4[idx];
    float4 inp_grad_val;
 
    if (means){
      xhat_f4.x = (inp_val.x - means[blockIdx.x]) * rsqrtf(vars[blockIdx.x]);
      xhat_f4.y = (inp_val.y - means[blockIdx.x]) * rsqrtf(vars[blockIdx.x]);
      xhat_f4.z = (inp_val.z - means[blockIdx.x]) * rsqrtf(vars[blockIdx.x]);
      xhat_f4.w = (inp_val.w - means[blockIdx.x]) * rsqrtf(vars[blockIdx.x]);

    } else {
      xhat_f4.x = (inp_val.x - betta_val.x) / gamma_val.x;
      xhat_f4.y = (inp_val.y - betta_val.y) / gamma_val.y;
      xhat_f4.z = (inp_val.z - betta_val.z) / gamma_val.z;
      xhat_f4.w = (inp_val.w - betta_val.w) / gamma_val.w;
    }

    dxhat_f4.x = out_grad_val.x * gamma_val.x; 
    dxhat_f4.y = out_grad_val.y * gamma_val.y; 
    dxhat_f4.z = out_grad_val.z * gamma_val.z; 
    dxhat_f4.w = out_grad_val.w * gamma_val.w; 

    inp_grad_val.x = (dxhat_f4.x - (s_sum_dxhat + (xhat_f4.x * s_sum_dxhat_xhat)) / (hidden_dim * 4)) * rsqrtf(vars[blockIdx.x]);
    inp_grad_val.y = (dxhat_f4.y - (s_sum_dxhat + (xhat_f4.y * s_sum_dxhat_xhat)) / (hidden_dim * 4)) * rsqrtf(vars[blockIdx.x]);
    inp_grad_val.z = (dxhat_f4.z - (s_sum_dxhat + (xhat_f4.z * s_sum_dxhat_xhat)) / (hidden_dim * 4)) * rsqrtf(vars[blockIdx.x]);
    inp_grad_val.w = (dxhat_f4.w - (s_sum_dxhat + (xhat_f4.w * s_sum_dxhat_xhat)) / (hidden_dim * 4)) * rsqrtf(vars[blockIdx.x]);
  
    inp_grad_f4[idx] = inp_grad_val;
  }
  /// END ASSIGN4_2_2
}

extern "C" {
void launch_layernorm_bw(float *gamma_grad, float *betta_grad, float *inp_grad,
                         const float *out_grad, const float *inp, const float *gamma,
                         const float *betta, const float *vars,
                         const float *means, int batch_size, int hidden_dim,
                         hipStream_t stream_1, hipStream_t stream_2) {
  
  // Allocate device memory
  float *d_gamma_grad, *d_betta_grad, *d_inp_grad, *d_out_grad, *d_inp, *d_gamma, *d_betta, *d_vars, *d_means;
  int grad_output_size = batch_size * hidden_dim * sizeof(float);
  int gamma_betta_size = hidden_dim * sizeof(float);
  int vars_means_size = batch_size * sizeof(float);

  hipMalloc((void **)&d_gamma_grad, gamma_betta_size);
  hipMalloc((void **)&d_betta_grad, gamma_betta_size);
  hipMalloc((void **)&d_inp_grad, grad_output_size);
  hipMalloc((void **)&d_out_grad, grad_output_size);
  hipMalloc((void **)&d_inp, grad_output_size);
  hipMalloc((void **)&d_gamma, gamma_betta_size);
  hipMalloc((void **)&d_betta, gamma_betta_size);
  hipMalloc((void **)&d_vars, vars_means_size);
  hipMalloc((void **)&d_means, vars_means_size);

  // Copy memory to device
  hipMemcpy((void *)d_out_grad, out_grad, grad_output_size, hipMemcpyHostToDevice);
  hipMemcpy((void *)d_inp, inp, grad_output_size, hipMemcpyHostToDevice);
  hipMemcpy((void *)d_gamma, gamma, gamma_betta_size, hipMemcpyHostToDevice);
  hipMemcpy((void *)d_betta, betta, gamma_betta_size, hipMemcpyHostToDevice);
  hipMemcpy((void *)d_vars, vars, vars_means_size, hipMemcpyHostToDevice);
  hipMemcpy((void *)d_means, means, vars_means_size, hipMemcpyHostToDevice);

  // Launch kernels
  // Compute grad of gamma and betta
  // This calculates the number of blocks needed to cover the data along the specified dimension, rounds it up.
  // The result is then multiplied by TILE_DIM to ensure that the grid size is a multiple of TILE_DIM.
  dim3 grid_dim(((hidden_dim + TILE_DIM - 1) / TILE_DIM) * TILE_DIM);
  dim3 block_dim(TILE_DIM, TILE_DIM);
  ker_ln_bw_dgamma_dbetta<float><<<grid_dim, block_dim, 0, stream_1>>>(
      d_gamma_grad, d_betta_grad, d_out_grad, d_inp, d_gamma, d_betta, d_vars,
      d_means, batch_size, hidden_dim);

  // Compute grad of input
  if (hidden_dim % 4 != 0 || hidden_dim > 4096) {
    throw std::runtime_error("hidden_dim % 4 != 0 || hidden_dim > 4096");
  }
  hidden_dim >>= 2;
  int nthread = min(((hidden_dim + 31) / 32) * 32, MAX_THREADS);
  ker_ln_bw_dinp<<<batch_size, nthread, 0, stream_2>>>(
      d_inp_grad, d_out_grad, d_inp, d_gamma, d_betta, d_vars, d_means, hidden_dim);

  // Synchronize and check for errors
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "launch_layernorm_bw Error: %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Copy back to host
  hipMemcpy(gamma_grad, d_gamma_grad, gamma_betta_size, hipMemcpyDeviceToHost);
  hipMemcpy(betta_grad, d_betta_grad, gamma_betta_size, hipMemcpyDeviceToHost);
  hipMemcpy(inp_grad, d_inp_grad, grad_output_size, hipMemcpyDeviceToHost);

  // Free device memory
  hipFree(d_gamma_grad);
  hipFree(d_betta_grad);
  hipFree(d_inp_grad);
  hipFree((void *)d_out_grad);
  hipFree((void *)d_inp);
  hipFree((void *)d_gamma);
  hipFree((void *)d_betta);
  hipFree((void *)d_vars);
  hipFree((void *)d_means);
}}
}}
